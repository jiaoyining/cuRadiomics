

#define _USE_MATH_DEFINES
#include "CVIPtexture.h"

void  RadiomicsCalculator_rl(const int *image, float *texture, const int *SET, int batch_size, int size0, int size1){

	int Range[] = {SET[0], SET[1]};
	//int Range[] = {0, 255};
	int BW = BIN_WIDTH;
	int Ng = (SET[1] - SET[0] + 1) / BW;

	int choose[] = {SET[2], SET[3]};
	int MASK_VALUE = SET[4];

	float Epsilon = EPSILON;
	int size[] = {size0, size1};
	int stride[] = {size1, 1};
	int n_batch = batch_size;
	//int MASK_VALUE = mask_value[0];


	// Constant Values concerning Directions of GLCM
	int NA = 4;
	int angles[] = {
		1, 1,
		1, 0,
		1, -1,
		0, 1};

	//printf("%d， %d, %d, %d, %d, %d", SET[0], SET[1], SET[2], SET[3], SET[4], SET[5]);
	if (choose[0] == 1)
	{
		PROPERTY_glcm Property_glcm;

	    (&Property_glcm)->P = Calculate_GLCM_rl(image, size, stride, angles, Range, MASK_VALUE, BW, Ng, NA, n_batch);
		//(&Property_fo)->P = Calculate_firstorder_rl(image, size, stride, bin, Binwidth, n_batch);
		//printf("glcm caculated! \n");

		//printf("memory allocated! \n");
		Calculate_GLCM_Property(&Property_glcm, Epsilon, Ng, NA, n_batch);
		//Calculate_firstorder_Property(&Property_fo, Epsilon, bin, Binwidth, Ng,  n_batch);
		//printf("property_glcm calculated! \n");

		Calculate_GLCM_Texture_rl(&Property_glcm, texture, Epsilon, Ng, NA, n_batch);
		//Calculate_firstorder_Texture_rl(&Property_fo, &texture[23 * n_batch], Epsilon, bin, Ng, Binwidth, n_batch);
		hipFree(Property_glcm.P);
		hipFree(Property_glcm.Pn);

		hipFree(Property_glcm.Px);
		hipFree(Property_glcm.Py);
		hipFree(Property_glcm.s);
		hipFree(Property_glcm.ux);
		hipFree(Property_glcm.uy);
		hipFree(Property_glcm.Dx);
		hipFree(Property_glcm.Dy);
		hipFree(Property_glcm.Pxay);

		hipFree(Property_glcm.Pxsy);
		hipFree(Property_glcm.HX);
		hipFree(Property_glcm.HY);
		hipFree(Property_glcm.HXY);
		hipFree(Property_glcm.HXY1);
		hipFree(Property_glcm.HXY2);
		hipFree(Property_glcm.DA);
		hipFree(Property_glcm.maxp);
	}

	if (choose[1] == 1)
	{

	PROPERTY_fo Property_fo;
	(&Property_fo)->P = Calculate_firstorder_rl(image, Range, MASK_VALUE, size, stride, Ng, BW, n_batch);
	Calculate_firstorder_Property(&Property_fo, Epsilon, BW, Ng, n_batch);
	Calculate_firstorder_Texture_rl(&Property_fo, &texture[23 * n_batch * choose[0]], Epsilon, Ng, BW, n_batch);

	hipFree(Property_fo.P);
	hipFree(Property_fo.Np);
	hipFree(Property_fo.Pn);

	hipFree(Property_fo.pn);
	hipFree(Property_fo.Pf);
	hipFree(Property_fo.PF);
	hipFree(Property_fo.P25);
	hipFree(Property_fo.P50);
	hipFree(Property_fo.P75);
	hipFree(Property_fo.P90);
	hipFree(Property_fo.P10);

	hipFree(Property_fo.Pmin);
	hipFree(Property_fo.Pmax);
	hipFree(Property_fo.Pm);
	hipFree(Property_fo.Pv);
	hipFree(Property_fo.N1090);
	hipFree(Property_fo.mP1090);
	}

	//printf("Texture_firstorder Calculated! \n");

	//texture_glcm = out_texture_glcm;

	//Property_glcm.~PROPERTY_glcm();
	//hipFree(glcm);

	//FreeProperty_glcm(Property_glcm);
	//hipFree(out_texture_glcm);


	//free(Property_glcm);
	//free(size);
	//free(angles);
	//free(stride);


	//hipFree(maxp);



/*
	hipFree(Property_glcm->P);
	hipFree(Property_glcm->Pn);

	hipFree(Property_glcm->Px);
	hipFree(Property_glcm->Py);
	hipFree(Property_glcm->s);
	hipFree(Property_glcm->ux);
	hipFree(Property_glcm->uy);
	hipFree(Property_glcm->Dx);
	hipFree(Property_glcm->Dy);
	hipFree(Property_glcm->Pxay);

	hipFree(Property_glcm->Pxsy);
	hipFree(Property_glcm->HX);
	hipFree(Property_glcm->HY);
	hipFree(Property_glcm->HXY);
	hipFree(Property_glcm->HXY1);
	hipFree(Property_glcm->HXY2);
	hipFree(Property_glcm->maxp);
	*/





	//printf("deleted! \n");
	//delete []Property_glcm;

	//free(Property_glcm);
	//free(glcm);

}















